#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <stdio.h>
// defining minimum and maximum sizes 
#define minSize 1024
#define maxSize 8192

using namespace std;

// function to print the current state
void StatePrint(int *a,long SIZE)
{
  for (unsigned y = 1; y < SIZE - 1; y++)
  {
    for (unsigned x = 1; x < SIZE - 1; x++)
    {
        a[y*SIZE+x] == 1 ? cout << char(219) << char(219) : cout << ' ' << ' ';
    }
    cout << endl;
  }
}

// function to build a random state in the grid
void randomGrid(int *a,long SIZE)
{
  srand(time(NULL));
  for (int y = 1; y < SIZE - 1; y++)
  {
    for (int x = 1; x < SIZE - 1; x++)
    {
      a[y*SIZE+x] = rand() % 2;
    }
  }
}

// function to add a blinker to the grid
void addBlinker(int *a, int i, int j,long SIZE)
{
  int b[][3] = {{0, 1, 0}, {0, 1, 0}, {0, 1, 0}};
  for (int p = 0; p < 3; p++)
  {
    for (int q = 0; q < 3; q++)
    {
 a[(i + p)*SIZE+j + q] = b[p][q];
    }
  }
}

// function to add a Glider to the grid at some coordinates
void addGlider(int *a, int i, int j,long SIZE)
{
  int b[][3] = {{0, 0, 1},
                {1, 0, 1},
                {0, 1, 1}};
  for (int p = 0; p < 3; p++)
  {
    for (int q = 0; q < 3; q++)
    {
      a[(i + p)*SIZE+j + q] = b[p][q];
    }
  }
}

// function to add a Glider gun to the grid at some coordinates
void addGliderGun(int *a, int i, int j,long SIZE)
{
  int b[11][38] = {0};
  b[5][1] = b[5][2] = 1;
  b[6][1] = b[6][2] = 1;

  b[3][13] = b[3][14] = 1;
  b[4][12] = b[4][16] = 1;
  b[5][11] = b[5][17] = 1;
  b[6][11] = b[6][15] = b[6][17] = b[6][18] = 1;
  b[7][11] = b[7][17] = 1;
  b[8][12] = b[8][16] = 1;
  b[9][13] = b[9][14] = 1;

  b[1][25] = 1;
  b[2][23] = b[2][25] = 1;
  b[3][21] = b[3][22] = 1;
  b[4][21] = b[4][22] = 1;
  b[5][21] = b[5][22] = 1;
  b[6][23] = b[6][25] = 1;
  b[7][25] = 1;
b[3][35] = b[3][36] = 1;
  b[4][35] = b[4][36] = 1;

  for (int p = 0; p < 11; p++)
  {
    for (int q = 0; q < 38; q++)
    {
      a[(i + p)*SIZE+j + q] = b[p][q];
    }
  }
}

// Defining kernel function to simulate cellular automata 
__global__ void cellular_automata(int *a,int *b,long SIZE)
{
  int count = 0;
  // number of new states to be generated
  int loop=100;
  // getting thread id
  long int tid=blockIdx.x*blockDim.x+threadIdx.x;
  long int row,col;
  //figuring out row id and column id 
  row=int(tid/SIZE);
  col=tid%SIZE;
  
  // evaluating a cell 
  while(loop){
    for (int i = -1; i < 2; i++)
    {
      for (int j = -1; j < 2; j++)
      {
        if (i != 0 || j != 0)
          count += (a[(row + i)*SIZE +col + j] ? 1 : 0);
      }
    }
    b[row*SIZE+col] = a[row*SIZE+col] == 1 ? count == 3 || count == 2 ? 1 : 0 : count == 3 ? 1 : 0;

    loop--;
    
    a[row*SIZE+col] = b[row*SIZE+col];
  }
}

int main()
{
  int *a;
  
  int *d_a, *d_b;

  float e_time1,e_time2,e_time3,milliseconds,throughput;

  FILE *data=fopen("parallel_data.txt","w");

  for(long SIZE=minSize;SIZE<=maxSize;SIZE*=2)
  {
    // allocating space for grid 
    a=(int*)malloc(SIZE*SIZE*sizeof(int));

    // initialising grid with a random state
    randomGrid(a,SIZE);
    //addGlider(a, 100, 100);
    //addGliderGun(a, 225, 100);
    //addBlinker(a, 125, 130);
    //StatePrint(a);

    // allocating memory in cuda device
    hipMalloc((void**)&d_a,SIZE*SIZE*sizeof(int));
    hipMalloc((void**)&d_b,SIZE*SIZE*sizeof(int));

    // creating events to record different timings
    hipEvent_t start1,stop1,start2,stop2,start3,stop3;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    // copying memory from host to device
    hipEventRecord(start1);
    hipMemcpy(d_a,a,SIZE*SIZE*sizeof(int),hipMemcpyHostToDevice);
    hipEventRecord(stop1);

    // calculating memory copy time
    hipEventSynchronize(stop1);
    milliseconds=0;
    hipEventElapsedTime(&milliseconds, start1, stop1);
    e_time1=(double)milliseconds/1000;
    cout<<"HTOD:"<<e_time1<<endl;

    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    // running the simulation
    hipEventRecord(start2);
    for(int j=0;j<(maxSize/SIZE);j++)
    {
      cellular_automata<<<SIZE*SIZE/1024,1024>>>(d_a,d_b,SIZE);
    }
    hipEventRecord(stop2);

    //calculating the compute time
    hipEventSynchronize(stop2);
    milliseconds=0;
    hipEventElapsedTime(&milliseconds, start2, stop2);
    e_time2=(double)milliseconds/1000;
    cout<<"Kernel:"<<e_time2<<endl;

    hipEventCreate(&start3);
    hipEventCreate(&stop3);

    // copying back the results of the simulation from device to host
    hipEventRecord(start3);
    hipMemcpy(a,d_a,SIZE*SIZE*sizeof(int),hipMemcpyDeviceToHost);
    hipEventRecord(stop3);

    // calculating memory copy time
    hipEventSynchronize(stop3);
    milliseconds=0;
    hipEventElapsedTime(&milliseconds, start3, stop3);
    e_time3=(double)milliseconds/1000;
    cout<<"DTOH:"<<e_time3<<endl;

    // calculating throughput
    throughput=(sizeof(float)*maxSize*maxSize)/e_time2;

    fprintf(data,"%lf,%lf,%lf,%lf\n",e_time1,e_time2*SIZE/maxSize,e_time3,throughput/1000000);
    
    // deallocating cuda device memory
    hipFree(d_a);
    hipFree(d_b);

    // deallocating grid space
    free(a);
  }

  cout<<"ENDED";
  return 0;
  // End of the Program
}
              